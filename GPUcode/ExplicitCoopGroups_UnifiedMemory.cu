#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h> 
#include <hip/hip_runtime.h> 
#include <hip/hip_cooperative_groups.h>
#include ""
#include <math.h>
#include<vector>
#include <iostream>
#include <stdio.h>
#include <assert.h>
#include <time.h>
#include <stdlib.h>
using namespace cooperative_groups;

struct BS_DiffEq { // struct for storing the BS params
    double _ak, _bk, _ck, _dk, _dt, _intRate;
    long long int _NS, _NT;
};


// can be wrapped around any runtime API call. No-op in release builds.
inline
hipError_t checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
    if (result != hipSuccess) {
        fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
        assert(result == hipSuccess);
    }
#endif
    return result;
}


__global__
void grscheme(double* _S, double* V, BS_DiffEq* pbs_diff_eq) // calculate BS by explicit FDM kernel
{

    int th = threadIdx.x; //thread number in block
    int blk_sz = blockDim.x; //block size
    int blk_id = blockIdx.x; //block number in grid
    int index = blk_sz * blk_id + th; // global index
    extern __shared__ double shdV[];  //dynamic shared memory array 
    auto g = this_grid(); // grid handle 
    double _ak = pbs_diff_eq->_ak;
    double _bk = pbs_diff_eq->_bk;
    double _ck = pbs_diff_eq->_ck;
    double _dk = pbs_diff_eq->_dk;
    double _dt = pbs_diff_eq->_dt;
    double _intRate = pbs_diff_eq->_intRate;
    long long int N_S = pbs_diff_eq->_NS; //stock levels
    long long int N_t = pbs_diff_eq->_NT; // time steps
    int thp1 = th + 1;
    if (index < N_S) { //conditional for fitting FD column size with GPU grid
        for (long long int k = 1; k < N_t; k++) //time stepping inside kernel
        {		
            shdV[thp1] = V[index + (k - 1) * N_S];  // each thread stores its current value
            shdV[0] = 0.0; // initialize lower neighbour
            shdV[blk_sz + 1] = 0.0; // initialize upper neighbour

            if (blk_id > 0) // if not in first block
            {
                //lower neighbour comes from lower neighbouring block
                shdV[0] = V[((k - 1) * N_S) + blk_id * blk_sz - 1];
            }


            if (blk_id < (gridDim.x - 1)) // if not in last block
            {
                //upper neighbour comes from upper neighbouring block
                shdV[blk_sz + 1] = V[(blk_id + 1) * blk_sz + ((k - 1) * N_S)];
            }


            double s = shdV[thp1], sm1 = 0.0, sp1 = 0.0; //define FD stencil

            __syncthreads(); //wait for all threads in block to gather their values

            if ((index - 1) > 0)
            {
                sm1 = shdV[thp1 - 1]; //read level i-1 from shared memory

            }
            
            if ((index + 1) < N_S)
            {
                sp1 = shdV[thp1 + 1]; //read level i+1 from shared memory
            }

            double Delta = _ak * (sp1 - sm1); //calculate delta=dV/dS
            double Gamma = _bk * (sp1 - 2 * s + sm1); //calculate gamma=dV/dS^2
            double Theta = _ck * _S[index] * _S[index] * Gamma - _intRate * _S[index] * Delta + _intRate * s; //calculate theta=dV/dt
            V[index + k * N_S] = V[index + (k - 1) * N_S] - Theta * _dt; //calculate V^k+1 from theta

            if (index == 0) //is first row
            {
                //First row tridiagonal matrix - dense vector product
                V[index + k * N_S] = _dk * V[index + (k - 1) * N_S];
            }

            if (index == N_S - 1) //is last row
            {
                //Last row tridiagonal matrix - dense vector product
                V[index + k * N_S] = 2 * V[N_S - 2 + k * N_S] - V[N_S - 3 + k * N_S];
            }    
            g.sync(); //synchronize full grid 
        }
    }
}

void solve_BS(int gs,double epn,int bs)
{
    // set problem parameters


    double Vol = 0.2, Int_Rate = 0.05, Expiration = epn, Strike = 100.0; //params BS
    //printf("Number of points per time step:");
    int block_size = bs;
    long long  int N_S=gs; //number of stock levels 
    //scanf("%d", &N_S);
    double dS = 2 * Strike / N_S; //asset step
    double dt = 0.9f / (Vol * Vol) / (N_S * N_S); //time step
    long long int N_t = ceil(Expiration / dt) + 1; //number of time steps

    printf("%lld\n", N_t);
    clock_t t0, t1, t2; //timing variables
    double t1sum = 0.0; //timing sum
    double t2sum = 0.0; //timing sum
#define V(I,J) V_mat[I + N_S * J] 

    dt = Expiration / N_t; //time step for calculations
    t0 = clock(); // initialize setup
    double *S,*V_mat; //unified stock price array, V matrix
    BS_DiffEq *pbs_diff_eq; //unified struct BS params
    
    //Unified memory allocations
    hipMallocManaged(&S, N_S*sizeof(double));
    hipMallocManaged(&V_mat, N_S * N_t * sizeof(double));
    hipMallocManaged(&pbs_diff_eq, sizeof(BS_DiffEq));
    int numBlocks = (N_S + block_size - 1) / block_size;

    S[0] = 10.0f;  //initial stock price
    for (int i = 1; i < N_S; i++) { // fill in stock value array
        S[i] = S[i - 1] + dS;
    }



    // set initial condition
    for (int i = 0; i < N_S; i++) { //initial V^k array
        V(i, 0) = fmaxf(S[i] - Strike, 0.0); //payoff function
    }
    printf("%8.3f\n", V_mat[N_S-1]);
    // evaluate coefficients that are needed in finite difference approximation
    double ak = (1 / (2 * dS));
    double bk = (1 / dS / dS);
    double ck = (-0.5 * Vol * Vol);
    double dk = (1 - Int_Rate * dt);

    pbs_diff_eq->_ak = ak;
    pbs_diff_eq->_bk = bk;
    pbs_diff_eq->_ck = ck;
    pbs_diff_eq->_dk = dk;
    pbs_diff_eq->_dt = dt;
    pbs_diff_eq->_intRate = Int_Rate;
    pbs_diff_eq->_NS = N_S;
    pbs_diff_eq->_NT = N_t;
   
    void * args[] = { (void*)&S, (void*)&V_mat, (void*)&pbs_diff_eq}; //argument array for cooperative kernel
    t1 = clock();  //setup time
    t1sum = ((double)(t1 - t0)) / CLOCKS_PER_SEC;
    printf("Init took %f seconds.  Begin compute\n", t1sum);
    hipLaunchCooperativeKernel((void*)grscheme, numBlocks, block_size, args,(block_size + 2)*sizeof(double)); //launch BS explicit FDM cooperative kernel
    hipDeviceSynchronize();  //device synchronization after all calculations
    //checkCuda(hipMemcpy(V_mat, d_V, (N_S * 10) * sizeof(double), hipMemcpyDeviceToHost));
    t2 = clock(); // computation time of full solution
    t2sum = ((double)(t2 - t1)) / CLOCKS_PER_SEC;
    printf("Computing took %f seconds.  Finish to compute\n", t2sum);

    printf("%8.3f", V((N_S - 1),(N_t-1)));

    checkCuda(hipFree(S));
    checkCuda(hipFree(V_mat));
    checkCuda(hipFree(pbs_diff_eq));

    /*checkCuda(hipFree(d_S));
    checkCuda(hipFree(d_V));
    checkCuda(hipFree(d_pbs_diff_eq));
    //#endif
    free(S);
    free(V_mat);
    free(pbs_diff_eq);*/
}

int main(int argc, char **argv)
{
    int gridsize=atoi(argv[1]); //stock levels
    double epn=atof(argv[2]);
    int blksize=atoi(argv[3]); //block size
    try {
        solve_BS(gridsize,epn,blksize);
    }
    catch (std::runtime_error err) {
        std::cout << err.what() << std::endl;
    }
    return 0;
}

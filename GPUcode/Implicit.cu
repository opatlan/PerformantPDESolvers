#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h> // hipMalloc, hipMemcpy, etc.
#include <hipsparse.h>         // hipsparseSpMV
#include <stdio.h>            // printf
#include <stdlib.h>           // EXIT_FAILURE
#include "hip/hip_runtime.h"
#include ""
#include <math.h>
#include <iostream>
#include <string.h> 


#define CHECK_CUDA(func)                                                       \
{                                                                              \
    hipError_t status = (func);                                               \
    if (status != hipSuccess) {                                               \
        printf("CUDA API failed at line %d with error: %s (%d)\n",             \
               __LINE__, hipGetErrorString(status), status);                  \
        return EXIT_FAILURE;                                                   \
    }                                                                          \
}

#define CHECK_CUSPARSE(func)                                                   \
{                                                                              \
    hipsparseStatus_t status = (func);                                          \
    if (status != HIPSPARSE_STATUS_SUCCESS) {                                   \
        printf("CUSPARSE API failed at line %d with error: %s (%d)\n",         \
               __LINE__, hipsparseGetErrorString(status), status);              \
        return EXIT_FAILURE;                                                   \
    }                                                                          \
}

struct BS_DiffEq { //struct for parameters of BS
    double _nu1, _nu2, _dt, _sigma, _intRate;
    long long int _NS, _NT;
};

//function to calculate mean squared error
double mean(double *V, double Vf, double Vo, long long int N_S)
{
    double acum=0.0; //acumulator
    double mn=0.0; //mean 
    int inc=N_S/100;
    for (long long int i = 0; i < N_S; i+=inc)
    {
        if(i==0)
        {
            acum += Vo;    //Initial Value
        }

        acum+=V[i];
        
        if(i==N_S-1)
        {
            acum += Vf;    //Last Value
        }

    }
    mn=acum/100; //Normalize
    return mn;
}
__global__
void calc_coef(double* _S, double* A, double* B, double* C, BS_DiffEq* pbs_diff_eq)
{

    int th = threadIdx.x; //thread number in block
    int blk_sz = blockDim.x; //block size
    int blk_id = blockIdx.x; //block number in grid
    int index = blk_sz * blk_id + th; // global index
    double _nu1 = pbs_diff_eq->_nu1; // dt / dS^2
    double _nu2 = pbs_diff_eq->_nu2; // dt / dS
    double _dt = pbs_diff_eq->_dt; // time step size
    double _volatility = pbs_diff_eq->_sigma;
    double _intRate = pbs_diff_eq->_intRate;
    long int N_S = pbs_diff_eq->_NS; //number of stock levels
    long int N_t = pbs_diff_eq->_NT; //number of time levels
    double a_k = 0.5 * (_volatility * _volatility) * (_S[index] * _S[index]);
    double b_k = _intRate * _S[index];
    double c_k = -_intRate;
    double Ak = 0.0;
    double Bk = 0.0;
    double Ck = 0.0;
    if (index < N_S) //conditional for fitting FD column size with GPU grid
    {
        Ak = -_nu1 * a_k + 0.5 * _nu2 * b_k;
        Bk = 2*_nu1 * a_k - _dt * c_k;
        Ck = -_nu1 * a_k - 0.5 * _nu2 * b_k;

        A[index] = Ak;
        B[index] = 1 + Bk;
        C[index] = Ck;

        if (index == N_S - 1)  // lower boundary condition
        {
            A[index] = Ak - Ck;
            B[index] = 1 + Bk + 2 * Ck;
        }                   
    }
}

int main(int argc, char** argv)
//int main(void)
{
    // Host problem definition
    int gridsize = atoi(argv[1]);  // grid size (number of asset levels)
    double volatility = atof(argv[2]);
    double expiration = atof(argv[3]);
    int blksize = atoi(argv[4]); //block size
    int tstsize = atoi(argv[5]);  //number of time levels

    double Vol = volatility, Int_Rate = 0.05, Expiration = expiration, Strike = 100.0; //params of BS

    //double Vol = 0.2, Int_Rate = 0.05, Expiration = 1.0, Strike = 100.0;
    int block_size = blksize;
    long long int N_Sp = gridsize; // total number of asset levels
    long long int N_S = N_Sp - 2; // total number of asset levels in FD matrices without boundary elements

    clock_t t0, t1, t2; //timing variables
    double t1sum = 0.0; //timing sum
    double t2sum = 0.0; //timing sum

    double dS = (2 * Strike) / N_Sp; //asset step

    long long int N_t = tstsize;
    double dt = Expiration / N_t; //time step
    const int nrh = 1; // number of right hand sides in algebraic solver
    const float h_one = 1;
    const float h_zero = 0;
    size_t lworkInBytes = 0;
    char* d_work = NULL;


    const int A_num_rows = N_S;
    const int A_num_cols = N_S;
    const int A_nnz = (A_num_cols + 2 * (A_num_cols - 1));

    t0 = clock();
    double* hX = (double*)malloc(N_Sp * sizeof(*hX)); //host V^k array
    double* hY = (double*)malloc(A_num_cols * sizeof(*hY)); //host V^k+1 array
    double* hY_result = (double*)malloc(N_Sp * sizeof(*hY_result));

    double* S = (double*)malloc(N_Sp * sizeof(*S)); // host stock array
    double* A = (double*)malloc(N_S * sizeof(*A)); // host coefficient A array
    double* B = (double*)malloc(N_S * sizeof(*B)); // host coefficient B array
    double* C = (double*)malloc(N_S * sizeof(*C)); // host coefficient C array
    BS_DiffEq* pbs_diff_eq = (BS_DiffEq*)malloc(sizeof(*pbs_diff_eq)); // params structure

    double     alpha = 1.0f; // alpha in y= alpha *Ax + beta*y
    double     beta = 0.0f;  // beta in y= alpha *Ax + beta*y

    //--------------------------------------------------------------------------
    // Device memory management
    double* d_S; // device stock array
    double* d_A; // device coefficient A array
    double* d_B; // device coefficient B array
    double* d_C; // device coefficient C array
    BS_DiffEq* d_pbs_diff_eq;

    long long int* dA_csrOffsets, * dA_columns;
    double* dA_values, * dX, * dY, * dY_result; // device  V^k, V^k+1 arrays

    // memory allocation of all device arrays
    CHECK_CUDA(hipMalloc((void**)&d_S, N_S * sizeof(double)))
    CHECK_CUDA(hipMalloc((void**)&d_A, N_S * sizeof(double)))
    CHECK_CUDA(hipMalloc((void**)&d_B, N_S * sizeof(double)))
    CHECK_CUDA(hipMalloc((void**)&d_C, N_S * sizeof(double)))
    CHECK_CUDA(hipMalloc((void**)&d_pbs_diff_eq, N_S * sizeof(BS_DiffEq)))
    CHECK_CUDA(hipMalloc((void**)&dX, A_num_cols * sizeof(double)))
    CHECK_CUDA(hipMalloc((void**)&dY, A_num_rows * sizeof(double)))
    CHECK_CUDA(hipMalloc((void**)&dY_result, N_Sp * sizeof(double)))



        for (int i = 0; i < N_Sp; i++) { // fill in stock value array
            S[i] = i * dS;
        }
    
    printf("%lf\n", S[N_Sp - 1]);



    // set initial condition

    for (int i = 0; i < N_Sp; i++) {  //initial V^k array
        hX[i] = fmaxf(S[i] - Strike, 0.0); //payoff function
    }

    printf("%lf\n", hX[N_Sp - 1]);
 
    double nu1 = (dt / (dS * dS)); // dt / dS^2
    double nu2 = (dt / dS); // dt / dS

    pbs_diff_eq->_nu1 = nu1;
    pbs_diff_eq->_nu2 = nu2;
    pbs_diff_eq->_dt = dt;
    pbs_diff_eq->_sigma = Vol;
    pbs_diff_eq->_intRate = Int_Rate;
    pbs_diff_eq->_NS = N_S;
    pbs_diff_eq->_NT = N_t;

    int numBlocks = (N_S + block_size - 1) / block_size; //number of blocks 
    CHECK_CUDA(hipMemcpy(d_pbs_diff_eq, pbs_diff_eq, sizeof(BS_DiffEq), hipMemcpyHostToDevice))
        CHECK_CUDA(hipMemcpy(dX, &hX[1], N_S * sizeof(double), hipMemcpyHostToDevice))
        CHECK_CUDA(hipMemcpy(d_S, &S[1], N_S * sizeof(double), hipMemcpyHostToDevice))
        CHECK_CUDA(hipMemset(d_A, 0, N_S * sizeof(double)))
        CHECK_CUDA(hipMemset(d_B, 0, N_S * sizeof(double)))
        CHECK_CUDA(hipMemset(d_C, 0, N_S * sizeof(double)))
        CHECK_CUDA(hipMemcpy(dY, &hX[1], N_S * sizeof(double), hipMemcpyHostToDevice))

    hipsparseHandle_t     handle = NULL; //handle to cuSPARSE
    CHECK_CUSPARSE(hipsparseCreate(&handle)) //cuSPARSE matrix descriptor


    double V_o = 0.0; // first value in V^k array (upper boundary condition)
    double V_lo = hX[0]; // first value in V^k array first time step
    double* V_fi = (double*)malloc(2*sizeof(double)); // two final values in V^k array required for computing  the lower boundary condition
    double V_f = 0.0; // last value in V^k array (upper boundary condition)
    t1 = clock(); //setup time
    t1sum = ((double)(t1 - t0)) / CLOCKS_PER_SEC;
    printf("Init took %f seconds.  Begin compute\n", t1sum);
    //launch calculate coefficients kernel
    calc_coef << <numBlocks, block_size >> > (d_S, d_A, d_B, d_C, d_pbs_diff_eq);
    //device syncrhonization after kernel execution
    hipDeviceSynchronize();

        CHECK_CUSPARSE(hipsparseDgtsv2_nopivot_bufferSizeExt(
            handle, N_S,
            nrh, d_A, d_B, d_C, dY, N_S,
            &lworkInBytes))
        CHECK_CUDA(hipMalloc((void**)&d_work, lworkInBytes))
        
    for (int i = 0; i < N_t; i++) //time step loop
    {


        //solve tridiagonal system using CR-PCR algorithm
            CHECK_CUSPARSE(hipsparseDgtsv2_nopivot(
                handle, N_S,
                nrh, d_A, d_B, d_C, dY, N_S,
                d_work))
                
            CHECK_CUDA(hipDeviceSynchronize())
            
        V_o = V_lo * (1 - Int_Rate * dt); //Calculate upper boundary condition
        V_lo = V_o; // update first value for next iteration
        //--------------------------------------------------------------------------
        // device result check
        //CHECK_CUDA(hipMemcpy(dX, dY, A_num_rows * sizeof(double), hipMemcpyDeviceToDevice))
            CHECK_CUDA(hipMemcpy(V_fi, &dY[N_S-2], 2 * sizeof(double), hipMemcpyDeviceToHost))
	    V_f = 2 * V_fi[1] - V_fi[0]; // calculate lower boundary conditions

        //CHECK_CUDA(hipMemset(dY, 0, N_S * sizeof(double)))
    }

    t2 = clock(); // computation time of full solution
    t2sum = ((double)(t2 - t1)) / CLOCKS_PER_SEC;
    printf("Computing took %f seconds.  Finish to compute\n", t2sum);

    CHECK_CUDA(hipMemcpy(hY, dY, A_num_rows * sizeof(double), hipMemcpyDeviceToHost)) //copy solution

    printf("%lf\n", V_f); // print final value of V^k
    printf("\n");
    double mn=0.0; //initialize mean squared value variable
    mn=mean(hY,V_f,V_o,N_Sp); // call mean squared value function
    printf("%lf\n",mn); // print mean squared value
    printf("End\n"); // print END

    //--------------------------------------------------------------------------
    // 
    // device memory deallocation
    CHECK_CUSPARSE(hipsparseDestroy(handle))
    CHECK_CUDA(hipFree(d_work))
        CHECK_CUDA(hipFree(dX))
        CHECK_CUDA(hipFree(dY))
        CHECK_CUDA(hipFree(dY_result))
        CHECK_CUDA(hipFree(d_A))
        CHECK_CUDA(hipFree(d_B))
        CHECK_CUDA(hipFree(d_C))
        CHECK_CUDA(hipFree(d_pbs_diff_eq))

    // host memory deallocation
    free(S);
    free(A);
    free(B);
    free(C);
    free(hX);
    free(hY);
    free(hY_result);
    free(V_fi);
    free(pbs_diff_eq);
    return EXIT_SUCCESS;
}

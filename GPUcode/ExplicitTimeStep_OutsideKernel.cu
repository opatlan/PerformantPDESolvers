
#include <hip/hip_runtime.h>
#include <math.h>
#include <vector>
#include <iostream>
#include <stdio.h>
#include <assert.h>
#include <time.h>

struct BS_DiffEq { // struct for storing the BS params
    float _ak, _bk, _ck, _dk, _dt, _intRate;
    long long int _NS, _NT;
};

// can be wrapped around any runtime API call. No-op in release builds.
inline
hipError_t checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
    if (result != cudaSuccess) {
        fprintf(stderr, "CUDA Runtime Error: %s\n", cudaGetErrorString(result));
        assert(result == cudaSuccess);
    }
#endif
    return result;
}

__global__
void grscheme(float* _S, float* V, float* Vp, BS_DiffEq* pbs_diff_eq) // calculate BS by explicit FDM kernel
{

    int th = threadIdx.x; //thread number in block 
    int blk_sz = blockDim.x; //block size
    int blk_id = blockIdx.x;  //block number in grid
    int index = blk_sz * blk_id + th; // global index
    extern __shared__ float shdV[]; //dynamic shared memory array 

    float _ak = pbs_diff_eq->_ak;
    float _bk = pbs_diff_eq->_bk;
    float _ck = pbs_diff_eq->_ck;
    float _dk = pbs_diff_eq->_dk;
    float _dt = pbs_diff_eq->_dt;
    float _intRate = pbs_diff_eq->_intRate;
    long long int N_S = pbs_diff_eq->_NS; //stock levels
    long long int N_t = pbs_diff_eq->_NT; // time steps 
    int thp1 = th + 1;

    shdV[thp1] = V[index]; // each thread stores its current value
    shdV[0] = 0.0; // initialize lower neighbour
    shdV[blk_sz + 1] = 0.0; // initialize upper neighbour

    if (blk_id > 0) // if not in first block
    {
        //lower neighbour comes from lower neighbouring block
        shdV[0] = V[blk_id * blk_sz - 1];
    }


    if (blk_id < (gridDim.x - 1))  // if not in last block
    {
        //upper neighbour comes from upper neighbouring block
        shdV[blk_sz + 1] = V[(blk_id + 1) * blk_sz];
        
    }


    float s = shdV[thp1], sm1 = 0.0, sp1 = 0.0; //define FD stencil

    __syncthreads(); //wait for all threads in block to gather their values

 
    if ((index - 1) > 0)
    {
        sm1 = shdV[thp1 - 1]; //read level i-1 from shared memory
    }
    if ((index + 1) < N_S)
    {
        sp1 = shdV[thp1 + 1]; //read level i+1 from shared memory
    }

    float Delta = _ak * (sp1 - sm1);//(V[th + 1] - V[th - 1]); //calculate delta=dV/dS
    float Gamma = _bk * (sp1 - 2 * s + sm1); //V[th + 1] - 2 * V[th] + V[th - 1]); //calculate gamma=dV/dS^2
    float Theta = _ck * _S[index] * _S[index] * Gamma - _intRate * _S[index] * Delta + _intRate * s; //calculate theta=dV/dt
    Vp[index] = V[index] - Theta * _dt; //calculate V^k+1 from theta
    if (index == 0) //is first value in V^k 
    {
        Vp[index] = _dk * V[index]; //Calculate upper boundary condition
    } 
    if (index == N_S - 1) //is last value in V^k 
    {
        Vp[index] = 2 * Vp[N_S - 2] - Vp[N_S - 3]; //Calculate lower boundary condition
    }    
    __syncthreads(); //wait for all values to finish 
}

void solve_BS(int gs, int bs)
{
    // set problem parameters


    float Vol = 0.2, Int_Rate = 0.05, Expiration = 1.0, Strike = 100.0; //params BS
    const int block_size = bs; 
    int N_S=gs; //number of stock levels 
    //scanf("%d", &N_S);
    float dS = 2 * Strike / N_S; //asset step
    float dt = 0.9f / (Vol * Vol) / (N_S * N_S);  //time step
    long long int N_t = ceil(Expiration / dt) + 1; //number of time steps

    clock_t t0, t1, t2; //timing variables
    double t1sum = 0.0; //timing sum
    double t2sum = 0.0; //timing sum

    #define V(I,J) V_mat[I + N_S * J] 

    dt = Expiration / N_t; //time step for calculations

    t0 = clock(); // initialize setup
    // initialize stock price grid
    float* S = new float[N_S]; //host stock price array
    float* V_mat = new float[N_S * N_t]; //host V^k array
    BS_DiffEq* pbs_diff_eq = new BS_DiffEq; //host struct BS params

    float* d_S; //device stock price array
    float* d_V; //device V^k array
    float* d_Vp; //device V^k+1 array
    BS_DiffEq* d_pbs_diff_eq; //device struct BS params

    checkCuda(hipMalloc((void**)&d_S, N_S * sizeof(float)));
    checkCuda(hipMalloc((void**)&d_V, N_S * sizeof(float)));
    checkCuda(hipMalloc((void**)&d_Vp, N_S * sizeof(float)));
    checkCuda(hipMalloc((void**)&d_pbs_diff_eq, sizeof(BS_DiffEq)));

    S[0] = 10.0f; //initial stock price
    for (int i = 1; i < N_S; i++) { // fill in stock value array
        S[i] = S[i - 1] + dS;
    }

    for (int i = 0; i < N_S; i++) { //initial V^k array
        V(i, 0) = fmaxf(S[i] - Strike, 0.0); //payoff function
    }

    // evaluate coefficients that are needed in finite difference approximation
    float ak = (1 / (2 * dS));
    float bk = (1 / dS / dS);
    float ck = (-0.5 * Vol * Vol);
    float dk = (1 - Int_Rate * dt);

    pbs_diff_eq->_ak = ak;
    pbs_diff_eq->_bk = bk;
    pbs_diff_eq->_ck = ck;
    pbs_diff_eq->_dk = dk;
    pbs_diff_eq->_dt = dt;
    pbs_diff_eq->_intRate = Int_Rate;
    pbs_diff_eq->_NS = N_S;
    pbs_diff_eq->_NT = N_t;

    int numBlocks = (N_S + block_size - 1) / block_size; //number of blocks 

    checkCuda(hipMemcpy(d_S, S, N_S * sizeof(float), hipMemcpyHostToDevice));
    checkCuda(hipMemcpy(d_pbs_diff_eq, pbs_diff_eq, sizeof(BS_DiffEq), hipMemcpyHostToDevice));

    t1 = clock(); //setup time 
    t1sum = ((double)(t1 - t0)) / CLOCKS_PER_SEC;
    printf("Init took %f seconds.  Begin compute\n", t1sum);

    for (long long int k = 1; k < N_t; k++) //out kernel time step loop
    {
        checkCuda(hipMemcpy(d_V, &V(0, (k - 1)), N_S * sizeof(float), hipMemcpyHostToDevice));
        checkCuda(hipMemset(d_Vp, 0, N_S * sizeof(float)));
        grscheme <<<numBlocks, block_size, (block_size + 2)*sizeof(float)>>> (d_S, d_V, d_Vp, d_pbs_diff_eq); //dynamic shared memory kernel 
        checkCuda(hipMemcpy(&V(0, k), d_Vp, N_S * sizeof(float), hipMemcpyDeviceToHost)); //update V^k with V^k+1
    }
    hipDeviceSynchronize(); //device synchronization after all calculations

    t2 = clock(); // computation time of full solution
    t2sum = ((double)(t2 - t1)) / CLOCKS_PER_SEC;
    printf("Computing took %f seconds.  Finish to compute\n", t2sum);


    for (long long int k = 0; k < N_t - 1; k += ceil(N_t / 10) + 1)
    {
        printf("%8.3f", V((N_S - 1), k));
    }

    checkCuda(hipFree(d_S));
    checkCuda(hipFree(d_V));
    checkCuda(hipFree(d_Vp));
    checkCuda(hipFree(d_pbs_diff_eq));
    free(S);
    free(V_mat);
    free(pbs_diff_eq);
}

int main(int argc, char **argv)
{
    int gridsize=atoi(argv[1]); //stock levels
    int blksize=atoi(argv[2]); //block size
    try {
        solve_BS(gridsize,blksize);
    }
    catch (std::runtime_error err) {
        std::cout << err.what() << std::endl;
    }
    return 0;
}

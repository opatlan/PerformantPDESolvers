#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h> 
#include <hip/hip_runtime.h> 
#include <hip/hip_cooperative_groups.h>
#include ""
#include <math.h>
#include<vector>
#include <iostream>
#include <stdio.h>
#include <assert.h>
#include <time.h>
#include <stdlib.h>
using namespace cooperative_groups;

struct BS_DiffEq { // struct for storing the BS params
    double _ak, _bk, _ck, _dk, _dt, _intRate;
    long long int _NS, _NT;
};


// can be wrapped around any runtime API call. No-op in release builds.
inline
hipError_t checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
    if (result != hipSuccess) {
        fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
        assert(result == hipSuccess);
    }
#endif
    return result;
}


__global__
void grscheme(double* _S, double* V, double *Vp, BS_DiffEq* pbs_diff_eq) // calculate BS by explicit FDM kernel
{

    int th = threadIdx.x; //thread number in block
    int blk_sz = blockDim.x; //block size
    int blk_id = blockIdx.x; //block number in grid
    int index = blk_sz * blk_id + th; // global index
    extern __shared__ double shdV[]; //dynamic shared memory array
    auto g = this_grid(); // grid handle
    double _ak = pbs_diff_eq->_ak;
    double _bk = pbs_diff_eq->_bk;
    double _ck = pbs_diff_eq->_ck;
    double _dk = pbs_diff_eq->_dk;
    double _dt = pbs_diff_eq->_dt;
    double _intRate = pbs_diff_eq->_intRate;
    long long int N_S = pbs_diff_eq->_NS; //stock levels
    long long int N_t = pbs_diff_eq->_NT; // time steps
    int thp1 = th + 1;
    if (index < N_S) { //conditional for fitting FD column size with GPU grid
        for (long long int i = 1; i < N_t; i++) //time stepping inside kernel
        {
            
            shdV[thp1] = V[index]; // each thread stores its current value
            shdV[0] = 0.0; // initialize lower neighbour
            shdV[blk_sz + 1] = 0.0; // initialize upper neighbour

            if (blk_id > 0) // if not in first block
            {
                //lower neighbour comes from lower neighbouring block
                shdV[0] = V[blk_id * blk_sz - 1];
            }


            if (blk_id < (gridDim.x - 1)) // if not in last block
            {
                //upper neighbour comes from upper neighbouring block
                shdV[blk_sz + 1] = V[(blk_id + 1) * blk_sz];
                
            }


            double s = shdV[thp1], sm1 = 0.0, sp1 = 0.0; //define FD stencil
           
            __syncthreads(); //wait for all threads in block to gather their values
            if ((index - 1) > 0)
            {
                sm1 = shdV[thp1 - 1]; //read level i-1 from shared memory
            }
            if ((index + 1) < N_S)
            {
                sp1 = shdV[thp1 + 1]; //read level i+1 from shared memory
            }

            double Delta = _ak * (sp1 - sm1); //calculate delta=dV/dS
            double Gamma = _bk * (sp1 - 2 * s + sm1); //calculate gamma=dV/dS^2
            double Theta = _ck * _S[index] * _S[index] * Gamma - _intRate * _S[index] * Delta + _intRate * s; //calculate theta=dV/dt
            Vp[index] = V[index] - Theta * _dt; //calculate V^k+1 from theta
            if (index == 0) //is first row
            {
                //First row tridiagonal matrix - dense vector product
                Vp[index] = _dk * V[index];
            }
            if (index == N_S - 1) //is last row
            {
                //Last row tridiagonal matrix - dense vector product
                Vp[index] = 2 * Vp[N_S - 2] - Vp[N_S - 3];
            }    
           V[index] = Vp[index]; //Update V^k with V^k+1
           
           g.sync();  //synchronize full grid 
        }
    }
}

//function to calculate mean squared error
double mean(double *V, long long int N_S)
{
    double acum=0.0; //acumulator
    double mn=0.0; //mean 
    int inc=N_S/100;
    for (long long int i = 0; i < N_S; i+=inc)
    {
        acum+=V[i];

    }
    mn=acum/100; //Normalize
    return mn;
}

void solve_BS(int gs ,double vty ,double epn, int bs)
{
    // set problem parameters


    double Vol = vty, Int_Rate = 0.05, Expiration = epn, Strike = 100.0; //params BS
    int block_size = bs; 
    long long  int N_S = gs; //number of stock levels
    //scanf("%d", &N_S);
    double dS = 2 * Strike / N_S; //asset step
    double dt = 0.9f / (Vol * Vol) / (N_S * N_S); //time step
    long long int N_t = ceil(Expiration / dt) + 1;  //number of time steps
    clock_t t0, t1, t2; //timing variables
    double t1sum = 0.0; //timing sum
    double t2sum = 0.0; //timing sum
#define V(I,J) V_mat[I + N_S * J] 

    dt = Expiration / N_t; //time step for calculations

    t0 = clock(); // initialize setup
    // initialize stock price grid
    double* S = (double*)malloc(N_S * sizeof(*S)); //host stock price array
    double* V_mat = (double*)malloc(N_S * sizeof(*V_mat));  //host V^k array
    BS_DiffEq* pbs_diff_eq = (BS_DiffEq*)malloc(sizeof(*pbs_diff_eq)); //host struct BS params

    double* d_S; //device stock price array
    double* d_V; //device V^k array
    double* d_Vp; //device V^k+1 array
    BS_DiffEq* d_pbs_diff_eq; //device struct BS params

    
    checkCuda(hipMalloc((void**)&d_S, N_S * sizeof(double)));
    checkCuda(hipMalloc((void**)&d_V, N_S * sizeof(double)));
    checkCuda(hipMalloc((void**)&d_Vp, N_S * sizeof(double)));
    checkCuda(hipMalloc((void**)&d_pbs_diff_eq, sizeof(BS_DiffEq)));

    S[0] = 10.0f; //initial stock price
    for (int i = 1; i < N_S; i++) { // fill in stock value array
        S[i] = S[i - 1] + dS;
    }
    printf("%8.3f\n", S[N_S-1]);

    // set initial condition
    for (int i = 0; i < N_S; i++) { //initial V^k array
        V(i, 0) = fmaxf(S[i] - Strike, 0.0);  //payoff function
    }

    printf("%8.3f\n", V_mat[N_S-1]);
    // evaluate coefficients that are needed in finite difference approximation
    double ak = (1 / (2 * dS));
    double bk = (1 / dS / dS);
    double ck = (-0.5 * Vol * Vol);
    double dk = (1 - Int_Rate * dt);
    pbs_diff_eq->_ak = ak;
    pbs_diff_eq->_bk = bk;
    pbs_diff_eq->_ck = ck;
    pbs_diff_eq->_dk = dk;
    pbs_diff_eq->_dt = dt;
    pbs_diff_eq->_intRate = Int_Rate;
    pbs_diff_eq->_NS = N_S;
    pbs_diff_eq->_NT = N_t;

    int numBlocks = (N_S + block_size - 1) / block_size; //number of blocks
    checkCuda(hipMemcpy(d_S, S, N_S * sizeof(double), hipMemcpyHostToDevice));
    checkCuda(hipMemcpy(d_pbs_diff_eq, pbs_diff_eq, sizeof(BS_DiffEq), hipMemcpyHostToDevice));
    checkCuda(hipMemcpy(d_V, V_mat, N_S * sizeof(double), hipMemcpyHostToDevice));
    checkCuda(hipMemset(d_Vp, 0, N_S * sizeof(double)));
    //printf("%lld\n",N_t);
    void* args[] = { (void*)&d_S, (void*)&d_V,  (void*)&d_Vp,(void*)&d_pbs_diff_eq };  //argument array for cooperative kernel
    t1 = clock(); //setup time
    t1sum = ((double)(t1 - t0)) / CLOCKS_PER_SEC;
    printf("Init took %f seconds.  Begin compute\n", t1sum);
    hipLaunchCooperativeKernel((void*)grscheme, numBlocks, block_size, args, (block_size + 2) * sizeof(double)); //launch BS explicit FDM cooperative kerne
    hipDeviceSynchronize(); //device synchronization after all calculations
    checkCuda(hipMemcpy(V_mat, d_V, N_S  * sizeof(double), hipMemcpyDeviceToHost)); //copy solution
    t2 = clock(); // computation time of full solution
    t2sum = ((double)(t2 - t1)) / CLOCKS_PER_SEC;
    printf("Computing took %f seconds.  Finish to compute\n", t2sum);

   printf("%lf\n", V_mat[N_S-1]);
   double mn=0.0;
   mn=mean(V_mat,N_S);
   printf("%lf\n",mn);
   printf("End\n");

    checkCuda(hipFree(d_S));
    checkCuda(hipFree(d_V));
    checkCuda(hipFree(d_pbs_diff_eq));
    //#endif
    free(S);
    free(V_mat);
    free(pbs_diff_eq);
}

int main(int argc, char** argv)
{
    int gridsize = atoi(argv[1]); //stock levels
    double volatility=atof(argv[2]); //volatility
    double expiration=atof(argv[3]); //expiration
    int blksize=atoi(argv[4]); //block size
    //int tstsize=atoi(argv[5]);
    try {
        solve_BS(gridsize,volatility,expiration,blksize);
        //solve_BS(gridsize,volatility,expiration,blksize,tstsize);
    }
    catch (std::runtime_error err) {
        std::cout << err.what() << std::endl;
    }
    return 0;
}
